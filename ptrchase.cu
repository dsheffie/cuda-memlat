#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <sys/mman.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <iostream>
#include <chrono>
#include <cstring>
#include <cassert>
#include <fstream>

struct node {
  node *next;
};

template <typename T>
void swap(T &x, T &y) {
  T t = x;
  x = y; y = t;
}

template <typename T>
void shuffle(std::vector<T> &vec, size_t len) {
  for(size_t i = 0; i < len; i++) {
    size_t j = i + (rand() % (len - i));
    swap(vec[i], vec[j]);
  }
}

template <typename T>
size_t partition(T *arr, size_t n) {
  size_t d=0;
  size_t r = rand() % n;
  T p = arr[r];
  arr[r] = arr[n-1];
  arr[n-1] = p;
  
  for(size_t i=0;i<(n-1);i++) {
    if(arr[i] < p) {
      swap(arr[i], arr[d]);
      d++;
    }
  }
  arr[n-1] = arr[d];
  arr[d] = p;
  return d;
}

template <typename T>
void sort(T *arr, size_t len) {
  size_t d;
  if(len <= 16) {
    for(size_t i=1;i<len;i++) {
      size_t j=i;
      while((j > 0) && (arr[j-1] > arr[j])) {
	swap(arr[j-1], arr[j]);
	j--;
      }
    }    
    return;
  }
  d = partition(arr, len);
  sort(arr, d);
  sort(arr+d+1, len-d-1);
}


__global__ void traverse(node **nodes, int64_t *cycles, uint64_t iters) {
  int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
  node *n = nodes[idx];
  int64_t now = clock64();
  while(iters) {
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;
    n = n->next;    
    iters -= 32;
  }
  cycles[idx] = clock64() - now;
  nodes[idx] = n;
}

int main(int argc, char *argv[]) {
  static const int nthr = 32;
  static const uint64_t max_keys = 1UL<<25;
  static_assert(sizeof(node*)==8);
  std::vector<uint64_t> keys(max_keys);
  node *nodes = nullptr, **nodes_out = nullptr;
  int64_t *cycles = nullptr;
  
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  if(deviceProp.kernelExecTimeoutEnabled) {
    std::cout << "Warning : kernel timeout enabled (long runs will fail)\n";
  }
  double freq = deviceProp.clockRate * 1000.0;
  
  assert(hipMallocManaged((void**)&nodes, sizeof(node)*max_keys) == hipSuccess);
  assert(hipMallocManaged((void**)&nodes_out, sizeof(node*)*nthr) == hipSuccess);  
  assert(hipMallocManaged((void**)&cycles, sizeof(int64_t)*nthr) == hipSuccess);

  std::ofstream out("gpulat.csv");
  
  for(uint64_t n_keys = 1UL<<8; n_keys <= max_keys; n_keys *= 2) {
    for(uint64_t i = 0; i < n_keys; i++) {
      keys[i] = i;
    }
    shuffle(keys, n_keys);
    node *h = &nodes[keys[0]];
    node *c = h;  
    h->next = h;
    for(uint64_t i = 1; i < n_keys; i++) {
      node *n = &nodes[keys[i]];
      node *t = c->next;
      c->next = n;
      n->next = t;
      c = n;
    }
    uint64_t iters = n_keys*16;
    for(int i = 0; i < nthr; i++) {
      nodes_out[i] = h;
    }
    
    if(iters < (1UL<<20)) {
      iters = 1UL<<20;
    }
    
    traverse<<<nthr/32, 32>>>(nodes_out, cycles, iters);
    hipDeviceSynchronize();
    auto ce = hipGetLastError();
    if(ce != hipSuccess) {
      std::cerr << hipGetErrorString(ce) << "\n";
    }
    sort(cycles, nthr);
    double cpl = static_cast<double>(cycles[nthr/2]) / iters;
    double nspl = (cpl/freq) / (1e-9);
    std::cout << sizeof(node)*n_keys << " bytes, GPU cycles per load "
	      << cpl << ", nanosec per load " << nspl << " \n";
    
    out << (sizeof(node)*n_keys) << ","
	<< cpl << ","<<  nspl << "\n";
    out.flush();
  }
  hipFree(nodes);
  hipFree(nodes_out);
  hipFree(cycles);
  return 0;
}
